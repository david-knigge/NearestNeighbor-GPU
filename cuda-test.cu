
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];

}
void random_ints(int *a, int M){
    int i;
    for(i=0; i<M;++i)
    {
        a[i] = rand();
    }
}

int main(void) {
    int *a, *b, *c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(int);
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);
    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU with N blocks
    add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
