#include "hip/hip_runtime.h"
#include <array>
#include <vector>
#include <bitset>

#include <cstdint>
#include <iostream>
#include "./generate_data.cpp"

#define NW 8 // use bitvectors of d=NW*32 bits, example NW=8
#define THREADS_PER_BLOCK 2 // Number of threads per block
#define NUMBER_OF_THREADS 10

using std::uint32_t; // 32-bit unsigned integer used inside bitvector
// using std::size_t;   // unsigned integer for indices

int total_counter = 0;

// type for bitvector
typedef array<uint32_t, NW> bitvec_t;
typedef array<uint32_t, 2> compound_t;
// type for lists of bitvectors
typedef vector<bitvec_t> list_t;
typedef vector<compound_t> output_t;

// type for any function that takes a list_t by reference
typedef void(*callback_list_t)(output_t);

__global__ void nns_kernel(uint32_t *start_vec_id, uint32_t *vecs, uint32_t *ret_vec, uint32_t *vector_size, uint32_t *l_size, uint32_t *thres)
{
    uint32_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t prim_vec = *start_vec_id + thread_id;

    uint32_t vectorweight, k;

    if (prim_vec < *l_size)
    {
        for (uint32_t j = 0; j < prim_vec; j++){
            vectorweight = 0;
            for (k = 0; k < *vector_size; k++){
                vectorweight += __popc(vecs[*vector_size * prim_vec + k] ^ vecs[*vector_size * j + k]);
            }
            // ret_vec = binary array (with 1 = hit, 0 = miss)
            //printf("prim_vec: %d sec_vec: %d vecw:%d arrayid: %d\n", prim_vec, j, vectorweight, thread_id * *l_size + j);
            ret_vec[thread_id * *l_size + j] = (vectorweight < *thres);
            //printf("%d, %d, thres: %d bool: %d\n", vectorweight, ret_vec[thread_id * *l_size + j], *thres, (vectorweight < *thres));
        }
    }
}

__host__ void clearlist(output_t output) {
    for (uint32_t i = 0; i < output.size(); i++) {
        total_counter += 1;
        //printf("%d,", output[i][0]);
        //printf("%d\n", output[i][1]);
    }
}

void NSS(const list_t& L, uint32_t t, callback_list_t f)  {

    output_t output;
    bitvec_t *vecs;
    uint32_t *vec, *vecd, *vecsd, *ret_vecd, *ret_vec, *vec_size, *vecd_size,
        *l_sized, *l_size, *thres, *thresd;

    // Initialize Host memory for vectors
    vec = (uint32_t *)malloc(sizeof(uint32_t));
    vecs = (bitvec_t *)malloc(sizeof(bitvec_t) * L.size());
    ret_vec = (uint32_t *)calloc(L.size() *NUMBER_OF_THREADS, sizeof(uint32_t));
    vec_size = (uint32_t *)malloc(sizeof(uint32_t));
    l_size = (uint32_t *)malloc(sizeof(uint32_t));
    thres = (uint32_t *)malloc(sizeof(uint32_t));

    // Copy location of data in vector
    memcpy(vecs, L.data(), L.size() * sizeof(bitvec_t));

    // Set vector size
    *vec_size = L[0].size();
    *l_size = L.size();
    *thres = t;

    // Allocate device memory for needed data
    hipMalloc((void **)&vecd, sizeof(bitvec_t));
    hipMalloc((void **)&vecsd,*l_size * sizeof(bitvec_t));
    hipMalloc((void **)&ret_vecd, *l_size * NUMBER_OF_THREADS * sizeof(uint32_t));
    hipMalloc((void **)&vecd_size, sizeof(uint32_t));
    hipMalloc((void **)&l_sized, sizeof(uint32_t));
    hipMalloc((void **)&thresd, sizeof(uint32_t));

    // Store L in device memory
    hipMemcpy(vecsd, vecs, *l_size * sizeof(bitvec_t), hipMemcpyHostToDevice);

    // Store vector size in device memory
    hipMemcpy(vecd_size, vec_size, sizeof(uint32_t), hipMemcpyHostToDevice);

    // Store list size in device memory
    hipMemcpy(l_sized, l_size, sizeof(uint32_t), hipMemcpyHostToDevice);

    hipMemcpy(thresd, thres, sizeof(uint32_t), hipMemcpyHostToDevice);

    // start first iteration at vector with index 1
    *vec = 1;
    hipMemcpy(vecd, vec, sizeof(uint32_t), hipMemcpyHostToDevice);

    nns_kernel<<< (NUMBER_OF_THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(vecd, vecsd, ret_vecd, vecd_size, l_sized, thresd);

    hipMemcpy(ret_vec, ret_vecd, *l_size * NUMBER_OF_THREADS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    uint32_t j,prim_vec, sec_vec;
    int i;
    int iterations = *l_size;
    for (i = 1 + NUMBER_OF_THREADS; i < iterations; i = i + NUMBER_OF_THREADS){
        // Initialize device memory to write found weights to
        *vec = i;

        hipMemcpy(vecd, vec, sizeof(uint32_t), hipMemcpyHostToDevice);
        nns_kernel<<< (NUMBER_OF_THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(vecd, vecsd, ret_vecd, vecd_size, l_sized, thresd);

        for (j = 0; j < NUMBER_OF_THREADS; j++)
        {
            prim_vec = i - NUMBER_OF_THREADS + j;
            if (prim_vec < *l_size)
            {
                for (sec_vec = 0; sec_vec < prim_vec; sec_vec++) {
                    // check if hit or miss
                    //printf("%d: %d\n",prim_vec, sec_vec);
                    if(ret_vec[j * *l_size + sec_vec])
                    {
                        compound_t callback_pair;
                        callback_pair[0] = prim_vec;
                        callback_pair[1] = sec_vec;
                        output.emplace_back(callback_pair);
                    }

                }
            }
        }

        // Empty output list
        f(output);
        output.clear();

        // Retrieve found weights from GPU memory
        hipMemcpy(ret_vec, ret_vecd, *l_size * NUMBER_OF_THREADS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    }
    for (j = 0; j < NUMBER_OF_THREADS; j++)
    {
        prim_vec = i - NUMBER_OF_THREADS + j;
        if (prim_vec < *l_size)
        {
            for (sec_vec = 0; sec_vec < prim_vec; sec_vec++) {
                // check if hit or miss
                if(ret_vec[j * *l_size + sec_vec])
                {
                    compound_t callback_pair;
                    callback_pair[0] = prim_vec;
                    callback_pair[1] = sec_vec;
                    output.emplace_back(callback_pair);
                }

            }
        }
    }

    // Empty output list
    f(output);
    output.clear();

    hipFree(vecd); hipFree(vecsd); hipFree(ret_vecd); hipFree(vecd_size); hipFree(l_sized);
    hipFree(thresd);
    free(vec); free(ret_vec); free(vecs); free(vec_size); free(l_size); free(thres);
}

int main() {
    list_t test;
    uint32_t leng = 100000;

    clock_t start;
    double duration;
    start = clock();

    generate_random_list(test, leng);
    uint32_t t = 110;

    NSS(test, t, clearlist);

    duration = (clock() - start ) / (double) CLOCKS_PER_SEC;
    cout<<"printf: "<< duration <<'\n';

    cout << leng << '\n';
    cout << total_counter << '\n';
    cout.flush();
    return 0;
}
