#include "hip/hip_runtime.h"
#include <array>
#include <vector>
#include <bitset>

#include <cstdint>
#include <iostream>
#include "./generate_data.cpp"

#define NW 8 // use bitvectors of d=NW*32 bits, example NW=8
#define THREADS_PER_BLOCK 256 // Number of threads per block
#define NUMBER_OF_THREADS 1024

using std::uint32_t; // 32-bit unsigned integer used inside bitvector
// using std::size_t;   // unsigned integer for indices

// type for bitvector
typedef array<uint32_t, NW> bitvec_t;
typedef array<uint32_t, 2> compound_t;
// type for lists of bitvectors
typedef vector<bitvec_t> list_t;
typedef vector<compound_t> output_t;

// type for any function that takes a list_t by reference
typedef void(*callback_list_t)(output_t);

__global__ void nns_kernel(uint32_t *vec_id, uint32_t *vecs, uint32_t *ret_vec, uint32_t *vector_size, uint32_t *l_size, uint32_t *thres)
{
    // uint32_t threadID = threadIdx.x + blockIdx.x * blockDim.x;
    for (uint32_t j = 0; j<*vec_id; j++){
      uint32_t vectorweight = 0;
      // printf("ThreadID: %d prim_vec: %d sec_vec: %d\n",threadID, i, j);
      for (uint32_t k = 0; k < NW; k++){
        vectorweight += __popc(vecs[NW * j + k] ^ vecs[NW * *vec_id + k]);
      }
      // ret_vec = binary array (with 1 = hit, 0 = miss)
      ret_vec[*vec_id * *l_size + j] = (vectorweight < *thres) ;
    }
}

__host__ void clearlist(output_t output) {
    for (uint32_t i = 0; i < output.size(); i++) {
        //printf("%zu,", output[i][0]);
        //printf("%zu\n", output[i][1]);
    }
}

void NSS(const list_t& L, uint32_t t, callback_list_t f)  {

    output_t output;
    bitvec_t *vecs;
    uint32_t *vec, *vecd, *vecsd, *ret_vecd, *ret_vec, *vec_size, *vecd_size,
        *l_sized, *l_size, *thres, *thresd;

    // Initialize Host memory for vectors
    vec = (uint32_t *)malloc(sizeof(uint32_t));
    vecs = (bitvec_t *)malloc(sizeof(bitvec_t) * L.size());
    ret_vec = (uint32_t *)calloc(sizeof(uint32_t) * (L.size() *NUMBER_OF_THREADS), sizeof(uint32_t));
    vec_size = (uint32_t *)malloc(sizeof(uint32_t));
    l_size = (uint32_t *)malloc(sizeof(uint32_t));
    thres = (uint32_t *)malloc(sizeof(uint32_t));
    // Copy location of data in vector
    memcpy(vecs, L.data(), L.size() * sizeof(bitvec_t));

    // Set vector size
    *vec_size = L[0].size();
    *l_size = L.size();

    // Allocate device memory for needed data
    hipMalloc((void **)&vecd, sizeof(bitvec_t));
    hipMalloc((void **)&vecsd, L.size() * sizeof(bitvec_t));
    hipMalloc((void **)&ret_vecd, L.size() * NUMBER_OF_THREADS * sizeof(uint32_t));
    hipMalloc((void **)&vecd_size, sizeof(uint32_t));
    hipMalloc((void **)&l_sized, sizeof(uint32_t));
    hipMalloc((void **)&thresd, sizeof(uint32_t));

    // Store L in device memory
    hipMemcpy(vecsd, vecs, L.size() * sizeof(bitvec_t), hipMemcpyHostToDevice);

    // Store vector size in device memory
    hipMemcpy(vecd_size, vec_size, sizeof(uint32_t), hipMemcpyHostToDevice);

    // Store list size in device memory
    hipMemcpy(l_sized, l_size, sizeof(uint32_t), hipMemcpyHostToDevice);

    uint32_t i,j;
    uint32_t iterations = L.size() - NUMBER_OF_THREADS;
    for (i = 0; i <iterations; i = i+NUMBER_OF_THREADS){
      // Initialize device memory to write found weights to
      *vec = i;
      hipMemcpy(vecd, vec, sizeof(uint32_t), hipMemcpyHostToDevice);
      nns_kernel<<< (NUMBER_OF_THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(vecd, vecsd, ret_vecd, vecd_size, l_sized, thresd);
    }

    // Launch kernel
    // Retrieve found weights from GPU memory
    hipMemcpy(ret_vec, ret_vecd, L.size() * NUMBER_OF_THREADS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    for (i = 0; i < L.size(); ++i)
    {
      for (j = 0; j < NUMBER_OF_THREADS; j++)
      {
          // check if hit or miss
          if(ret_vec[i])
          {
            compound_t callback_pair;
            callback_pair[0] = j;
            callback_pair[1] = i;
            output.emplace_back(callback_pair);
          }
      }
    }
    // periodically give outputlist back for further processing
    f(output); // assume it empties output
    output.clear();

    hipFree(vecd); hipFree(vecsd); hipFree(ret_vecd); hipFree(vecd_size); hipFree(l_sized);
    hipFree(thresd);
    free(vec); free(ret_vec); free(vecs); free(vec_size); free(l_size); free(thres);
}

int main() {
    list_t test;
    uint32_t leng = 100000;

    clock_t start;
    double duration;
    start = clock();

    generate_random_list(test, leng);
    uint32_t t = 128;

    NSS(test, t, clearlist);

    duration = (clock() - start ) / (double) CLOCKS_PER_SEC;
    cout<<"printf: "<< duration <<'\n';

    cout << leng << '\n';
    cout.flush();
    return 0;
}
