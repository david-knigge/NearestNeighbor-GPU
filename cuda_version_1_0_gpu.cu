#include "hip/hip_runtime.h"
#include <array>
#include <vector>
#include <bitset>

#include <cstdint>
#include <iostream>
#include "./generate_data.cpp"

// VERSIE 1.1:
// - zonder threshold op de gpu

#define NW 8 // use bitvectors of d=NW*32 bits, example NW=8
#define THREADS_PER_BLOCK 512 // Number of threads per block

int total_counter = 0;
using std::uint32_t; // 32-bit unsigned integer used inside bitvector
// using std::size_t;   // unsigned integer for indices

// type for bitvector
typedef array<uint32_t, NW> bitvec_t;
typedef array<uint32_t, 2> compound_t;
// type for lists of bitvectors
typedef vector<bitvec_t> list_t;
typedef vector<compound_t> output_t;


// type for any function that takes a output_t by reference
typedef void(*callback_list_t)(output_t *);

// takes in two pointers to the address of two bitvec_t's and a third pointer to
// where the results need to go
__global__ void cuda_xor(uint32_t *vec_1, uint32_t *vecs, uint32_t *ret_vec) {

    // compute which vector the thread has to do the xor operation on
    uint32_t vectorindex = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t wordindex;
    // the variable in which the amount of ones after the xor are added
    uint32_t vectorweight;

    // make sure the vectorindex is within the amount of vectors
    if (vectorindex < *vec_1) {
        vectorweight = 0;

        /* for each word in the vector do the xor operation with the
         * corresponding word of the other vector and count the ones
         * with popc
        */
        for (wordindex = 0; wordindex < NW; ++wordindex) {
            vectorweight += __popc(vecs[(*vec_1 * NW) + wordindex] ^
                                    vecs[(vectorindex * NW) + wordindex]);
        }
        ret_vec[vectorindex] = vectorweight;
    }
}

// Takes an output list and prints the indices per line
void print_output(output_t *output) {
    for (uint32_t i = 0; i < (*output).size(); i++) {
        total_counter += 1;
        //printf("1: %d  ", output[i][0]);
        //printf("2: %d\n", output[i][1]);
    }
    (*output).clear();
}

// takes in a reference to vector full of bitvec_t, an uint32 for the threshold
// and a function for handling the output compares all the vectors in L and does
// Nearest neighbour search.
void NSS(const list_t& L, uint32_t t, callback_list_t f)  {

    output_t output;

    // allocate space for all the variable pointers needed
    bitvec_t *vecs;
    uint32_t *vec, *vecd, *vecsd, *ret_vecd, *ret_vec, *l_size;
    //int size = L.size() * sizeof(bitvec_t);
    int size = sizeof(bitvec_t);
    l_size = (uint32_t *)malloc(sizeof(uint32_t));
    *l_size = L.size();


    // allocate space for all the actual values
    vec = (uint32_t *)malloc(sizeof(uint32_t));
    vecs = (bitvec_t *)malloc(sizeof(bitvec_t) * L.size());
    ret_vec = (uint32_t *)calloc(L.size(), sizeof(uint32_t));

    memcpy(vecs, L.data(), *l_size * size);

    // Allocate space for device copies of our primary vector, our entire setup
    // of vectors
    hipMalloc((void **)&vecd, size);
    hipMalloc((void **)&vecsd, *l_size * size);
    hipMalloc((void **)&ret_vecd, *l_size * sizeof(uint32_t));
    hipMemcpy(vecsd, vecs, *l_size * size, hipMemcpyHostToDevice);

    // allocate space fo vector indices
    uint32_t i,j;

    *vec = 1;
    hipMemcpy(vecd, vec, sizeof(uint32_t), hipMemcpyHostToDevice);

    // run 1 kernel
    cuda_xor<<<1, THREADS_PER_BLOCK>>>(vecd, vecsd, ret_vecd);
    // collect the results by copying from the gpu back to the cpu
    hipMemcpy(ret_vec, ret_vecd, L.size() * sizeof(uint32_t),
                hipMemcpyDeviceToHost);

    for (i = 1; i < *l_size; ++i) {

        *vec = i;
        hipMemcpy(vecd, vec, sizeof(uint32_t), hipMemcpyHostToDevice);

        // apply the cuda_xor function to all the up to i vectors
        cuda_xor<<<(i + THREADS_PER_BLOCK) / THREADS_PER_BLOCK,
                    THREADS_PER_BLOCK>>>(vecd, vecsd, ret_vecd);

        for (j = 0; j < i - 1; j++) {
            if (ret_vec[j] < t) {
                // create a compound term to add to the output list
                compound_t callback_pair;
                callback_pair[0] = i;
                callback_pair[1] = j;
                output.emplace_back(callback_pair);
            }
        }
        // collect the results by copying from the gpu back to the cpu
        hipMemcpy(ret_vec, ret_vecd, *l_size * sizeof(uint32_t),
                    hipMemcpyDeviceToHost);

        // periodically give outputlist back for further processing
        f(&output);

    }

    for (j = 0; j < i - 1; j++) {
        if (ret_vec[j] < t) {
            // create a compound term to add to the output list
            compound_t callback_pair;
            callback_pair[0] = i;
            callback_pair[1] = j;
            output.emplace_back(callback_pair);
        }
    }

    // Empty output list
    f(&output);
    // free the allocated memmory
    hipFree(vecd); hipFree(vecsd); hipFree(ret_vecd);
    free(vec); free(ret_vec); free(vecs);
}

int main() {
    list_t test;
    uint32_t leng = 5000;

    // starting the timer
    clock_t start;
    double duration;
    start = clock();

    // generating the dataset
    generate_random_list(test, leng);
    // setting the threshold
    uint32_t t = 110;

    NSS(test, t, print_output);

    // end the timer
    duration = (clock() - start ) / (double) CLOCKS_PER_SEC;
    cout<<"execution duration: "<< duration <<'\n';
    cout<<"total pairs: " << total_counter << '\n';
    cout.flush();
    return 0;
}
