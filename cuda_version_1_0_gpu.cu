#include "hip/hip_runtime.h"
#include <array>
#include <vector>
#include <bitset>

#include <cstdint>
#include <iostream>
#include "./generate_data.cpp"

// VERSIE 1.1:
// - zonder threshold op de gpu

#define NW 8 // use bitvectors of d=NW*32 bits, example NW=8
#define THREADS_PER_BLOCK 265 // Number of threads per block

using std::uint32_t; // 32-bit unsigned integer used inside bitvector
// using std::size_t;   // unsigned integer for indices

// type for bitvector
typedef array<uint32_t, NW> bitvec_t;
typedef array<uint32_t, 2> compound_t;
// type for lists of bitvectors
typedef vector<bitvec_t> list_t;
typedef vector<compound_t> output_t;


// type for any function that takes a output_t by reference
typedef void(*callback_list_t)(output_t);

// takes in two pointers to the address of two bitvec_t's and a third pointer to
// where the results need to go
__global__ void cuda_xor(uint32_t *vec_1, uint32_t *vecs, uint32_t *ret_vec) {

    // compute which vector the thread has to do the xor operation on
    uint32_t vectorindex = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t wordindex;
    // the variable in which the amount of ones after the xor are added
    uint32_t vectorweight;

    // make sure the vectorindex is within the amount of vectors
    if (vectorindex < *vec_1) {
        vectorweight = 0;

        /* for each word in the vector do the xor operation with the
         * corresponding word of the other vector and count the ones
         * with popc
        */
        for (wordindex = 0; wordindex < NW; ++wordindex) {
            vectorweight += __popc(vecs[(*vec_1 * NW) + wordindex] ^
                                    vecs[(vectorindex * NW) + wordindex]);
        }
        ret_vec[vectorindex] = vectorweight;
    }
}

// Takes an output list and prints the indices per line
void print_output(output_t output) {
    for (uint32_t i = 0; i < output.size(); i++) {
        // printf("%d,", output[i][0]);
        // printf("%d\n", output[i][1]);
    }
    output.clear();
}

// takes in a reference to vector full of bitvec_t, an uint32 for the threshold
// and a function for handling the output compares all the vectors in L and does
// Nearest neighbour search.
void NSS(const list_t& L, uint32_t t, callback_list_t f)  {

    output_t output;

    // allocate space for all the variable pointers needed
    bitvec_t *vecs;
    uint32_t *vec, *vecd, *vecsd, *ret_vecd, *ret_vec, *ret_vec_zeroes;
    int size = sizeof(bitvec_t);
    l_size = (uint32_t *)malloc(sizeof(uint32_t));
    *l_size = L.size();


    // allocate space for all the actual values
    vec = (uint32_t *)malloc(sizeof(uint32_t));
    vecs = (bitvec_t *)malloc(size * *l_size);
    ret_vec = (uint32_t *)calloc(*l_size , sizeof(uint32_t));
    ret_vec_zeroes = (uint32_t *)calloc(*l_size, sizeof(uint32_t));

    memcpy(vecs, L.data(), *l_size * size);

    // Allocate space for device copies of our primary vector, our entire setup
    // of vectors
    hipMalloc((void **)&vecd, size);
    hipMalloc((void **)&vecsd, *l_size * size);
    hipMalloc((void **)&ret_vecd, *l_size * sizeof(uint32_t));
    hipMemcpy(vecsd, vecs, *l_size * size, hipMemcpyHostToDevice);

    // allocate space fo vector indices
    uint32_t i,j;

    *vec = 1;
    // move the values from the cpu to the gpu
    hipMemcpy(vecd, vec, sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(ret_vecd, ret_vec_zeroes, L.size() * sizeof(uint32_t),
                hipMemcpyHostToDevice);
    // run 1 kernel
    cuda_xor<<<1, THREADS_PER_BLOCK>>>(vecd, vecsd, ret_vecd);
    // collect the results by copying from the gpu back to the cpu
    hipMemcpy(ret_vec, ret_vecd, L.size() * sizeof(uint32_t),
                hipMemcpyDeviceToHost);

    for (i = 1; i < *l_size; ++i) {

        *vec = i;
        hipMemcpy(vecd, vec, sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(ret_vecd, ret_vec_zeroes, *l_size * sizeof(uint32_t),
                    hipMemcpyHostToDevice);

        // apply the cuda_xor function to all the up to i vectors
        cuda_xor<<<(i + THREADS_PER_BLOCK) / THREADS_PER_BLOCK,
                    THREADS_PER_BLOCK>>>(vecd, vecsd, ret_vecd);

        for (j = 0; j < i-1; j++) {
            if (ret_vec[j] < t) {
                // create a compound term to add to the output list
                compound_t callback_pair;
                callback_pair[0] = i;
                callback_pair[1] = j;
                output.emplace_back(callback_pair);
            }
        }
        // collect the results by copying from the gpu back to the cpu
        hipMemcpy(ret_vec, ret_vecd, *l_size * sizeof(uint32_t),
                    hipMemcpyDeviceToHost);

        // periodically give outputlist back for further processing
        f(output);
        output.clear(); // clear the output

    }
    // free the allocated memmory
    hipFree(vecd); hipFree(vecsd); hipFree(ret_vecd);
    free(vec); free(ret_vec); free(vecs); free(ret_vec_zeroes);
}

int main() {
    list_t test;
    uint32_t leng = 10000;

    clock_t start;
    double duration;
    start = clock();

    generate_random_list(test, leng);
    uint32_t thersh = 110;
    cout << leng, cout << '\n';

    NSS(test, thersh, print_output);

    duration = (clock() - start ) / (double) CLOCKS_PER_SEC;
    cout<<"printf: "<< duration <<'\n';

    cout << "klaar\n";
    cout.flush();
    return 0;
}
