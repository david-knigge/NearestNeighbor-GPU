#include "hip/hip_runtime.h"
#include <array>
#include <vector>
#include <bitset>

#include <cstdint>
#include <iostream>
#include "./generate_data.cpp"

// VERSIE 1.1:
// - zonder threshold op de gpu

#define NW 8 // use bitvectors of d=NW*32 bits, example NW=8
#define THREADS_PER_BLOCK 265 // Number of threads per block

int total_counter = 0;
using std::uint32_t; // 32-bit unsigned integer used inside bitvector
// using std::size_t;   // unsigned integer for indices

// type for bitvector
typedef array<uint32_t, NW> bitvec_t;
typedef array<uint32_t, 2> compound_t;
// type for lists of bitvectors
typedef vector<bitvec_t> list_t;
typedef vector<compound_t> output_t;

// type for any function that takes 2 indices
// typedef void(*callback_pair_t)(size_t, size_t);
// type for any function that takes a list_t by reference

typedef void(*callback_list_t)(output_t);

__global__ void cuda_xor(uint32_t *vec_1, uint32_t *vecs, uint32_t *ret_vec)
{
    uint32_t vectorindex = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t wordindex;
    uint32_t vectorweight;

    if (vectorindex < *vec_1)
    {
        vectorweight = 0;
        for (wordindex = 0; wordindex < NW; ++wordindex)
        {
            vectorweight += __popc(vecs[(*vec_1 * NW) + wordindex] ^ vecs[(vectorindex * NW) + wordindex]);
            //printf("prim_vec: %d comp_vec: %d prim_word: %d comp_word: %d  score: %d vec_1: %d vec_2: %d xor: %d popc: %d\n", *vec_1, vectorindex, (*vec_1 * NW) + wordindex, (vectorindex * NW) + wordindex, vectorweight, vecs[(*vec_1 * NW) + wordindex], vecs[(vectorindex * NW) + wordindex],vecs[(*vec_1 * NW) + wordindex] ^ vecs[(vectorindex * NW) + wordindex], __popc(vecs[(*vec_1 * NW) + wordindex] ^ vecs[(vectorindex * NW) + wordindex]));
        }
        ret_vec[vectorindex] = vectorweight;
    }
}

inline uint32_t hammingweight(uint32_t n) {
   return __builtin_popcount(n);
}

void printsomestuff(output_t output) {
    for (uint32_t i = 0; i < output.size(); i++) {
        total_counter += 1;
        //printf("%d,", output[i][0]);
        //printf("%d\n", output[i][1]);
    }
    output.clear();
}

void NSS(const list_t& L, uint32_t t, callback_list_t f)  {

    output_t output;

    // go over all unique pairs 0 <= j < i < L.size()
    bitvec_t *vecs;
    uint32_t *vec, *vecd, *vecsd, *ret_vecd, *ret_vec, *ret_vec_zeroes;
    //int size = L.size() * sizeof(bitvec_t);
    int size = sizeof(bitvec_t);

    vec = (uint32_t *)malloc(sizeof(uint32_t));
    vecs = (bitvec_t *)malloc(sizeof(bitvec_t) * L.size());
    ret_vec = (uint32_t *)calloc(L.size(), sizeof(uint32_t));
    ret_vec_zeroes = (uint32_t *)calloc(L.size(), sizeof(uint32_t));

    memcpy(vecs, L.data(), L.size() * sizeof(bitvec_t));

    // Allocate space for device copies of our primary vector, our entire setup
    // of vectors
    hipMalloc((void **)&vecd, size);
    hipMalloc((void **)&vecsd, L.size() * size);
    hipMalloc((void **)&ret_vecd, L.size() * sizeof(uint32_t));
    hipMemcpy(vecsd, vecs, L.size() * size, hipMemcpyHostToDevice);

    uint32_t i,j;

    *vec = 1;
    hipMemcpy(ret_vecd, ret_vec_zeroes, L.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(vecd, vec, sizeof(uint32_t), hipMemcpyHostToDevice);
    // run 1 kernel
    cuda_xor<<<1, THREADS_PER_BLOCK>>>(vecd, vecsd, ret_vecd);
    hipMemcpy(ret_vec, ret_vecd, L.size() * sizeof(uint32_t), hipMemcpyDeviceToHost);

    for (i = 1; i < L.size(); ++i)    {

        *vec = i;
        hipMemcpy(ret_vecd, ret_vec_zeroes, L.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(vecd, vec, sizeof(uint32_t), hipMemcpyHostToDevice);

        cuda_xor<<<(i + THREADS_PER_BLOCK) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(vecd, vecsd, ret_vecd);

        for (j = 0; j < i - 1; j++)
        {
            if (ret_vec[j] < t)
            {
                compound_t callback_pair;
                callback_pair[0] = i;
                callback_pair[1] = j;
                output.emplace_back(callback_pair);
            }
        }
        hipMemcpy(ret_vec, ret_vecd, L.size() * sizeof(uint32_t), hipMemcpyDeviceToHost);

        // periodically give outputlist back for further processing
        f(output); // assume it empties output
        output.clear();

    }
    for (j = 0; j < i - 1; j++)
    {
        if (ret_vec[j] < t)
        {
            compound_t callback_pair;
            callback_pair[0] = i;
            callback_pair[1] = j;
            output.emplace_back(callback_pair);
        }
    }
    f(output); // assume it empties output
    output.clear();
    hipFree(vecd); hipFree(vecsd); hipFree(ret_vecd);
    free(vec); free(ret_vec); free(vecs); free(ret_vec_zeroes);
}

int main() {
    list_t test;
    uint32_t leng = 5000;

    clock_t start;
    double duration;
    start = clock();

    generate_random_list(test, leng);
    uint32_t thersh = 110;
    cout << leng, cout << '\n';

    NSS(test, thersh, printsomestuff);

    duration = (clock() - start ) / (double) CLOCKS_PER_SEC;
    cout<<"printf: "<< duration <<'\n';
    cout<<total_counter << '\n';
    cout << "klaar\n";
    cout.flush();
    return 0;
}
